#include "hip/hip_runtime.h"
#include "HyWallCuda.h"
#include "HyWall.h"
#include "GlobalMemoryHandler.h"
#include "HybridComputing.h"
#include "CoreData.h"
#include "UserSettings.h"
#include "Solver.h"
namespace HyWall
{
    __global__ void K_set_settings(UserSettings settings_in)
    {
        HyCore::settings = settings_in;
    }

    __global__ void K_set_access_pitch(int ap)
    {
        HyCore::majorAccessPitch = ap;
    }

    __global__ void K_meta(void)
    {
        HyCore::MetaDataSet(&HyCore::settings);
    }

    void CopyCudaSymbols(void)
    {

        K_set_settings<<<1,1>>>(settings);
        K_set_access_pitch<<<1,1>>>(memory.localGpuPoints);
        K_meta<<<1,1>>>();
        //TODO: calls to hipMemcpyToSymbol for global pointers!!!!!
    }

    void ComputeGpuSolution(void)
    {
        std::cout << "TERMINATED from file " << __FILE__ << ", line " << __LINE__ << ": no cuda support yet." << std::endl;
        abort();
    }

    void InitGpuSolution(void)
    {
        std::cout << "TERMINATED from file " << __FILE__ << ", line " << __LINE__ << ": no cuda support yet." << std::endl;
        abort();
    }

    void BeginGpuSolution(void)
    {
        std::cout << "TERMINATED from file " << __FILE__ << ", line " << __LINE__ << ": no cuda support yet." << std::endl;
        abort();
    }
}
