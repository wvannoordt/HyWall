#include "hip/hip_runtime.h"
#include "HyWallCuda.h"
#include "HyWall.h"
#include "GlobalMemoryHandler.h"
#include "HybridComputing.h"
#include "CoreData.h"
#include "UserSettings.h"
namespace HyWall
{
    __global__ void K_set_settings(UserSettings settings_in)
    {
        HyCore::settings = settings_in;
    }

    __global__ void K_set_access_pitch(int ap)
    {
        HyCore::majorAccessPitch = ap;
    }

    void CopyCudaSymbols(void)
    {

        K_set_settings<<<1,1>>>(settings);
        K_set_access_pitch<<<1,1>>>(memory.localGpuPoints);
        //TODO: calls to hipMemcpyToSymbol for global pointers!!!!!
    }

    void InitGpuSolution(void)
    {
        std::cout << "TERMINATED from file " << __FILE__ << ", line " << __LINE__ << ": no cuda support yet." << std::endl;
        abort();
    }

    void BeginGpuSolution(void)
    {
        std::cout << "TERMINATED from file " << __FILE__ << ", line " << __LINE__ << ": no cuda support yet." << std::endl;
        abort();
    }
}
